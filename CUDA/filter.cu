#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
extern "C" {
#include "ppmFile.h"
}

__global__ void mean(int *dW, int *dH, int *dR, unsigned char *dInput, unsigned char *dOutput) {
    int i, j, x, y, chan, temp, num, offset;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
	for (chan = 0; chan < 3; chan++) {
		temp = 0;
		num = 0;
		for (y = j - (*dR); y <= j + (*dR); y++) {
	
			for (x = i - (*dR); x <= i + (*dR); x++) {
				if (x >= 0 && x < *dW && y >= 0 && y < *dH) {
					offset = (y * (*dW) + x) * 3 + chan;
					temp += dInput[offset];
					num++;
				}
			}
		}
		temp /= num;
		offset = (j * (*dW) + i) * 3 + chan;
		dOutput[offset] = temp;
	}
}

__global__ void median(int *dW, int *dH, int *dR, unsigned char *dInput, unsigned char *dOutput) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int chan, offset;
    unsigned char filterVector[9] = {0,0,0,0,0,0,0,0,0};
    if((j==0) || (i==0) || (j==*dH-1) || (i==*dW-1))
        dOutput[i*(*dW)+j] = 0; //Deal with boundry conditions
    else {
        for(chan = 0; chan < 3; chan++){
            int index = 0;
            for (int y = j - (*dR); y <= j + (*dR); y++){
                for (int x = i - (*dR); x <= i + (*dR); x++){
                    offset = (y * (*dW) + x) * 3 + chan;
                    filterVector[index++] = dInput[offset];
                }
            }

            //Sort the filter array. This gives the median of the pixels at filterVector[4]
            for (int m = 0; m < 9; m++) {
                for (int n = m + 1; n < 9; n++) {
                    if (filterVector[m] > filterVector[n]) { 
                        //Swap the variables.
                        unsigned char tmp = filterVector[m];
                        filterVector[m] = filterVector[n];
                        filterVector[n] = tmp;
                    }
                }
            }
            offset = (j * (*dW) + i) * 3 + chan;
            dOutput[offset] = filterVector[4];
        }
    }
}

int main (int argc, char *argv[]) {
    int w, h, r, temp;
    Image *inImage;
    Image *outImage;
    unsigned char* data;
    double time;
    clock_t begin, end;
	unsigned char *dInput;
    unsigned char *dOutput;
    int *dW, *dH, *dR;
    char choice[10];
    
    if (argc != 5) {
		printf("Incorrect input arguments. Should be: <r> <input>.ppm <output>.ppm <choice>\n");
        return 0;
    }
    
	
	r = atoi(argv[1]);
	inImage = ImageRead(argv[2]);
    strcpy(choice, argv[4]);
	w = inImage->width;
	h = inImage->height;
	data = inImage->data;
	printf("Using image: %s, width: %d, height: %d, blur radius: %d\n",argv[2],w,h,r);
    printf("Waiting for GPU ...\n");

    // Grids are based on image size with blocks of 32x32
    dim3 blockD(32, 32);
    dim3 gridD((w + blockD.x - 1) / blockD.x, (h + blockD.y - 1) / blockD.y);
    

    hipMalloc((void**)&dInput, sizeof(unsigned char*) * w * h * 3);
    hipMalloc((void**)&dOutput, sizeof(unsigned char*) * w * h * 3);
    hipMalloc((void**)&dW, sizeof(int*));
    hipMalloc((void**)&dH, sizeof(int*));
    hipMalloc((void**)&dR, sizeof(int*));
    

    hipMemcpy(dInput, data, w * h * 3, hipMemcpyHostToDevice);
    hipMemcpy(dW, &w, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dH, &h, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dR, &r, sizeof(int), hipMemcpyHostToDevice);
    
	printf("Blurring image ...\n");
    printf("Grid size: %d x %d\n", gridD.x, gridD.y);
	printf("Block size: %d x %d\n", blockD.x, blockD.y);
    printf("Total number of threads: %d\n", gridD.x * gridD.y * blockD.x * blockD.y);
    
    if(!strcmp(choice, "mean")){
        begin = clock();
        
        mean<<<gridD, blockD>>>(dW, dH, dR, dInput, dOutput);

        end = clock();
    }
    else if (!strcmp(choice, "median")){
        begin = clock();
        
        median<<<gridD, blockD>>>(dW, dH, dR, dInput, dOutput);

        end = clock();
    }
    else{
        printf("\n\n~~~~~Incorrect filter choice. Use 'mean' or 'median' as an argument~~~~~\n\n");
        exit(1);
    }
    
    outImage = ImageCreate(w, h); //Create output image
	printf("Processing complete, assembling image ...\n");
    
    temp = w * h * 3;
    hipMemcpy(outImage->data, dOutput, temp, hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
        printf("\n\n````ERROR : %s````\n\n", hipGetErrorString(error));
    else
        printf("\n\n````NO ERRORS````\n\n");
	
	// Write output image to file
	ImageWrite(outImage, argv[3]);
	printf("Output image created: %s\n", argv[3]);
    
    time = (double)(end-begin) / CLOCKS_PER_SEC;
    printf("Filter execution time: %e s\n", time);
    
	free(inImage->data);
	free(outImage->data);
    hipFree(dInput);
    hipFree(dOutput);
    hipFree(dW);
    hipFree(dH);
    hipFree(dR);
    
    return 0;
}
